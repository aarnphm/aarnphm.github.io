#include "hip/hip_runtime.h"
// Example 1: Basic Vector Addition
// Demonstrates: Basic kernel launch, thread indexing, memory management

#include "common.cuh"
#include <stdio.h>

__global__ void vector_add(const half *a, const half *b, half *c, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    c[idx] = __hadd(a[idx], b[idx]);
  }
}

// CPU reference implementation
void vector_add_cpu(const half *a, const half *b, half *c, int N) {
  for (int i = 0; i < N; i++) {
    c[i] = __hadd(a[i], b[i]);
  }
}

int main() {
  printf("=== Vector Addition ===\n");
  print_device_info();

  const int N = 1 << 20; // 1M elements
  const size_t bytes = N * sizeof(half);

  // Allocate host memory
  half *h_a = (half *)malloc(bytes);
  half *h_b = (half *)malloc(bytes);
  half *h_c = (half *)malloc(bytes);
  half *h_c_ref = (half *)malloc(bytes);

  // Initialize input arrays
  init_array(h_a, N, __float2half(10.0f));
  init_array(h_b, N, __float2half(10.0f));

  // Allocate device memory
  half *d_a, *d_b, *d_c;
  CUDA_CHECK(hipMalloc(&d_a, bytes));
  CUDA_CHECK(hipMalloc(&d_b, bytes));
  CUDA_CHECK(hipMalloc(&d_c, bytes));

  // Copy inputs to device
  CUDA_CHECK(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice));

  // Launch configuration
  int threads_per_block = 256;
  int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;

  printf("Launch config: %d blocks × %d threads = %d total threads\n",
         blocks_per_grid, threads_per_block, blocks_per_grid * threads_per_block);

  // Launch kernel
  GpuTimer timer;
  timer.start();
  vector_add<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_c, N);
  timer.stop();
  CUDA_CHECK(hipGetLastError());

  // Copy result back
  CUDA_CHECK(hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost));

  printf("Kernel execution time: %.3f ms\n", timer.elapsed());
  printf("Throughput: %.2f GB/s\n",
         (3 * bytes) / (timer.elapsed() * 1e6)); // 2 reads + 1 write

  // Verify result
  vector_add_cpu(h_a, h_b, h_c_ref, N);
  bool correct = verify_results(h_c, h_c_ref, N, __float2half(1e-3f));
  printf("Verification: %s\n", correct ? "PASSED" : "FAILED");

  // Cleanup
  free(h_a);
  free(h_b);
  free(h_c);
  free(h_c_ref);
  CUDA_CHECK(hipFree(d_a));
  CUDA_CHECK(hipFree(d_b));
  CUDA_CHECK(hipFree(d_c));

  return 0;
}
